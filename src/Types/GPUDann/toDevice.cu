#include "./GPUDann.cuh"
#include "../../error/error.hpp"

template <>
void Cuno::GPUDann<double>::toDevice(
  double **layers,
  double **biases,
  double **weights,
  double **gradients,
  double **errors
) {
  if (!(this->valid)) {
    return;
  }

  for (int i = 0; i < this->length; i++) {
    hipMemcpy(
      this->layers[i], (double*)layers[i],
      sizeof(double) * this->arch[i],
      hipMemcpyHostToDevice
    );

    // SKIP OTHERS 
    if (i >= this->length-1) continue;

    hipMemcpy(
      this->biases[i], (double*)biases[i],
      sizeof(double) * this->arch[i+1],
      hipMemcpyHostToDevice
    );

    hipMemcpy(
      this->weights[i], weights[i],
      sizeof(double) * this->arch[i] * this->arch[i+1],
      hipMemcpyHostToDevice
    );

    hipMemcpy(
      this->gradients[i], (double*)gradients[i],
      sizeof(double) * this->arch[i+1],
      hipMemcpyHostToDevice
    );

    hipMemcpy(
      this->errors[i], (double*)errors[i],
      sizeof(double) * this->arch[i+1],
      hipMemcpyHostToDevice
    );
  }
} 
