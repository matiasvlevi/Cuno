#include "./MethodInput.cuh"
#include <hip/hip_runtime.h>

namespace Cuno {

template<>
void MethodInput<double>::allocate() {
  // TODO: CHECK FOR AVAILABLE SPACE IN DEVICE MEM

  hipMalloc(&(this->a), sizeof(double) * M * N); 
  hipMalloc(&(this->b), sizeof(double) * N * P);
  hipMalloc(&(this->c), sizeof(double) * M * P);
}

template<>
void MethodInput<double>::toDevice(double *values_a, double *values_b) {
  hipMemcpy(
      this->a, values_a,
      sizeof(double) * this->M * this->N,
      hipMemcpyHostToDevice
  );
  hipMemcpy(
      this->b, values_b,
      sizeof(double) * this->N * this->P,
      hipMemcpyHostToDevice
  );
}

template<>
void MethodInput<double>::getOutput(double *buffer) {
  hipMemcpy(
    buffer, this->c,
    sizeof(double) * this->M * this->P,
    hipMemcpyDeviceToHost
  );
} 

};