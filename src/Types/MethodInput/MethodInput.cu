#include "./MethodInput.cuh"
#include "../../error/error.hpp"
#include <hip/hip_runtime.h>

namespace Cuno {

template<>
bool MethodInput<double>::allocate() {
  // TODO: CHECK FOR AVAILABLE SPACE IN DEVICE MEM
  hipError_t error;
  
  error = hipMalloc(&(this->a), sizeof(double) * M * N); 
  if (error != hipSuccess) {
    Error::throw_("Matrix 'a' failed to properly allocate ");
    return false;
  }
  error = hipMalloc(&(this->b), sizeof(double) * N * P);
  if (error != hipSuccess) {
    Error::throw_("Matrix 'b' failed to properly allocate ");
    return false;
  }
  error = hipMalloc(&(this->c), sizeof(double) * M * P);
  if (error != hipSuccess) {
    Error::throw_("Matrix 'c' (result) failed to properly allocate ");
    return false;
  }

  return true;
}

template<>
void MethodInput<double>::toDevice(double *values_a, double *values_b) {
  if (!(this->valid)) return;
  hipMemcpy(
      this->a, values_a,
      sizeof(double) * this->M * this->N,
      hipMemcpyHostToDevice
  );
  hipMemcpy(
      this->b, values_b,
      sizeof(double) * this->N * this->P,
      hipMemcpyHostToDevice
  );
}

template<>
void MethodInput<double>::getOutput(double *buffer) {
  if (!(this->valid)) return;
   hipMemcpy(
    buffer, this->c,
    sizeof(double) * this->M * this->P,
    hipMemcpyDeviceToHost
  );
} 


};