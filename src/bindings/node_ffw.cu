#include "./bindings.cuh"

/**
  Wrapper for the batch train node binding, 
  creates a device dann instance, which allocates memory on the cuda device.
  Launches a cuda kernel after allocation
*/  
void Cuno::Bindings::FeedForward(
  const v8::FunctionCallbackInfo<v8::Value>& args
) {
  v8::Isolate* env = args.GetIsolate();
  v8::Local<v8::Context> context = env->GetCurrentContext();

  Cuno::GPUDann<double> *nn = 
  Cuno::v8Utils::FromNativeModel<double>(context, env, args); 

  double inputs[nn->arch[0]];

  for (int i = 0; i < nn->arch[0]; i++)
    inputs[i] = 0;
  

  Wrappers::ffw(nn, inputs);

  // from Device
  double buffer[1 * nn->arch[nn->length-1]];
  hipMemcpy(
      buffer, nn->layers[nn->length-1],
      nn->arch[nn->length-1] * sizeof(double),
      hipMemcpyDeviceToHost
  );

  // to v8
  v8::Local<v8::Array> output = 
    v8Utils::toJaggedArray<double>(context, env, buffer, 1, nn->arch[nn->length-1]); 

  // Wait for user input
  int x;
  std::cin >> x;

  // return v8 value
  args.GetReturnValue().Set(output);
}

