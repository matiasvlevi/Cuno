#include "../kernel.cuh"

void Kernel::TrainWrapper(
    ModelData *host,
    DeviceModelData *device
) {

  // Allocate memory 
  for (int i = 0; i < host->arch.size(); i++) { 
    hipMalloc(&(device->layers[i]), host->arch[i] * sizeof(double));
  }

  for (int i = 1; i < host->arch.size(); i++) {
    hipMalloc(&(device->biases[i-1]), host->arch[i] * sizeof(double));
    hipMalloc(&(device->errors[i-1]), host->arch[i] * sizeof(double));
    hipMalloc(&(device->gradients[i-1]), host->arch[i] * sizeof(double));
    hipMalloc(&(device->weights[i-1]), host->arch[i-1] * host->arch[i] * sizeof(double));
  }

  // Copy data in devices's input layer
  hipMemcpy(
    device->layers[0],
    host->inputs[0],
    host->arch[0] * sizeof(double),
    hipMemcpyHostToDevice
  );
 
  // Copy biases & weights to the device
  for (int i = 0; i < device->arch.size()-1; i++) {
    hipMemcpy(
        device->biases[i], host->biases[i],
        host->arch[i] * sizeof(double),
        hipMemcpyHostToDevice
    );

    hipMemcpy(
        device->weights[i], host->weights[i],
        host->arch[i] * host->arch[i+1] * sizeof(double),
        hipMemcpyHostToDevice
    );
  }

  // Kernel --- 

  dim3 THREADS;
  THREADS.x = 32;
  THREADS.y = 32;

  int blocks = (host->arch[1] + THREADS.y - 1) / THREADS.y;

  dim3 BLOCKS;
  BLOCKS.x = blocks;
  BLOCKS.y = blocks;

  Kernel::dotOpt<<<BLOCKS, THREADS>>>(
      device->weights[0], device->layers[0],
      device->layers[1],
      host->arch[0],
      host->arch[1], 1
  );

  // ---

  // Copy result back to host
  hipMemcpy(
    host->layers[1], device->layers[1],
    host->arch[1] * sizeof(double),
    hipMemcpyDeviceToHost
  );

  // Log debug
  host->logLayers();

  // deallocate layers
  for (int i = 0; i < device->arch.size(); i++) { 
    hipFree(device->layers[i]);
  }

  // deallocate other values
  for (int i = 0; i < device->arch.size(); i++) {
    hipFree(device->biases[i]);
    hipFree(device->errors[i]);
    hipFree(device->gradients[i]);
    hipFree(device->weights[i]);
  }

}
